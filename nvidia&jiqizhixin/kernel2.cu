#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <opencv2\opencv.hpp>
#include <iostream>
using namespace std;
using namespace cv;

float3 data_kernel[] = {
        make_float3(-1.0f, -1.0f, -1.0f), make_float3(0.0f, 0.0f, 0.0f), make_float3(1.0f, 1.0f, 1.0f),
        make_float3(-2.0f, -2.0f, -2.0f), make_float3(0.0f, 0.0f, 0.0f), make_float3(2.0f, 2.0f, 2.0f),
        make_float3(-1.0f, -1.0f, -1.0f), make_float3(0.0f, 0.0f, 0.0f), make_float3(1.0f, 1.0f, 1.0f),
        make_float3(-1.0f, -1.0f, -1.0f), make_float3(0.0f, 0.0f, 0.0f), make_float3(1.0f, 1.0f, 1.0f),
        make_float3(-2.0f, -2.0f, -2.0f), make_float3(0.0f, 0.0f, 0.0f), make_float3(2.0f, 2.0f, 2.0f),
        make_float3(-1.0f, -1.0f, -1.0f), make_float3(0.0f, 0.0f, 0.0f), make_float3(1.0f, 1.0f, 1.0f),
        make_float3(-1.0f, -1.0f, -1.0f), make_float3(0.0f, 0.0f, 0.0f), make_float3(1.0f, 1.0f, 1.0f),
        make_float3(-2.0f, -2.0f, -2.0f), make_float3(0.0f, 0.0f, 0.0f), make_float3(2.0f, 2.0f, 2.0f),
        make_float3(-1.0f, -1.0f, -1.0f), make_float3(0.0f, 0.0f, 0.0f), make_float3(1.0f, 1.0f, 1.0f)
};

int main()
{
    
    //利用opencv的接口读取图片,并取得相关信息：c, h, w...
    Mat img = imread("1.jpg");
    int imgWidth = img.cols;
    int imgHeight = img.rows;
    int imgChannel = img.channels();

    Mat dst_gpu(imgHeight, imgWidth, CV_8UC3, Scalar(0,0,0));   

    size_t num = imgChannel * imgHeight * imgWidth * sizeof(unsigned char);

    //1在GPU上分配空间
    unsigned char* in_gpu;
    unsigned char* out_gpu;
    float* filt_data;
    hipMalloc((void**)&filt_data, 3 * 3 * 3 * sizeof(float3));
    hipMalloc((void**)&in_gpu, num);
    hipMalloc((void**)&out_gpu, num);

    //2初始化句柄
    hipdnnHandle_t handle;
    hipdnnCreate(&handle);

    //3描述Tensor
    //input descriptor
    hipdnnTensorDescriptor_t input_descriptor;
    hipdnnCreateTensorDescriptor(&input_descriptor);
    hipdnnSetTensor4dDescriptor(input_descriptor, HIPDNN_TENSOR_NHWC, 
        HIPDNN_DATA_FLOAT, 1,3,imgHeight,imgWidth);

    //output descriptor
    hipdnnTensorDescriptor_t output_descriptor;
    hipdnnCreateTensorDescriptor(&output_descriptor);
    hipdnnSetTensor4dDescriptor(output_descriptor, HIPDNN_TENSOR_NHWC,
        HIPDNN_DATA_FLOAT, 1, 3, imgHeight, imgWidth);
    //kernel descriptor
    hipdnnFilterDescriptor_t kernel_descriptor;
    hipdnnCreateFilterDescriptor(&kernel_descriptor);
    hipdnnSetFilter4dDescriptor(kernel_descriptor, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 3,3,3,3);

        
    //4描述操作并设置相关参数
    hipdnnConvolutionDescriptor_t conv_descriptor;
    hipdnnCreateConvolutionDescriptor(&conv_descriptor);
    hipdnnSetConvolution2dDescriptor(conv_descriptor,
        1, 1, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

    
    //5描述算法
    hipdnnConvolutionFwdAlgoPerf_t algo;
    cudnnGetConvolutionForwardAlgorithm_v7(handle, input_descriptor,
        kernel_descriptor, conv_descriptor, output_descriptor, 1, 0, &algo);


   
        
    //6申请工作空间
    size_t workspace_size = 0;
    hipdnnGetConvolutionForwardWorkspaceSize(handle, input_descriptor,
        kernel_descriptor, conv_descriptor, output_descriptor, algo.algo, &workspace_size);

    void* workspace = nullptr;
    hipMalloc(&workspace, workspace_size);

        
    //7将计算需要的数据传输到GPU
    hipMemcpy((void*)filt_data, (void*)data_kernel, 3 * 3 * 3 * sizeof(float3),
        hipMemcpyHostToDevice);
    hipMemcpy(in_gpu, img.data, num, hipMemcpyHostToDevice);

    //8开始计算
    auto alpha = 1.0f, beta = 0.0f;
    hipdnnConvolutionForward(handle, &alpha, input_descriptor, in_gpu,
        kernel_descriptor, filt_data, conv_descriptor, algo.algo,
        workspace, workspace_size, &beta, output_descriptor, out_gpu);


        
    //9将计算结果传回CPU内存
    hipMemcpy(dst_gpu.data, out_gpu, num, hipMemcpyDeviceToHost);
        
    //10释放资源
    hipFree(in_gpu);
    hipFree(out_gpu);
    hipFree(workspace);

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(conv_descriptor);

    hipdnnDestroy(handle);
    imshow("cudnn_sample", dst_gpu);
    waitKey(0);
    
    return 0;
}
